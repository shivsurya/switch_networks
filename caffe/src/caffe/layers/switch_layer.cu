#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/switch_layer.hpp"
#include "caffe/layers/cudnn_pooling_layer.hpp"
#include "caffe/layers/pooling_layer.hpp"
#include "caffe/layers/conv_layer.hpp"
#include "caffe/layers/cudnn_conv_layer.hpp"
#include "caffe/layers/cudnn_lrn_layer.hpp"
#include "caffe/layers/lrn_layer.hpp"
#include "caffe/layers/im2col_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
void SwitchLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const int selector_ind = bottom.size() - 1;
  if(switch_type_==1)
  { 
  Dtype* top_data = top[0]->mutable_gpu_data();

  for (int n = 0; n < bottom[selector_ind]->num(); n++) {
    int index = static_cast<int>(bottom[selector_ind]->data_at(n, 0 , 0, 0));
    DCHECK(floor(index) == index) << "Index should be an integer";
    DCHECK_GE(index, 0) << "Index should be greater than 0";
    DCHECK_LT(index, selector_ind)
        << "Index should be less than " << selector_ind;
    const Dtype* bottom_data = bottom[index]->gpu_data();
    caffe_copy(num_elem_, bottom_data+bottom[index]->offset(n),
          top_data+top[0]->offset(n));
  }
  }
  else
  {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  vector<int> count_top_num(top.size());  // keeps count of the top_num

  for (int n = 0; n < bottom[selector_ind]->num(); n++) {
    int index = static_cast<int>(bottom[selector_ind]->data_at(n, 0 , 0, 0));

    Dtype* top_data = top[index]->mutable_gpu_data();
    int top_offset = count_top_num[index];
    count_top_num[index]++;


    caffe_copy(num_elem_, bottom_data + bottom[0]->offset(n),
        top_data + top[index]->offset(top_offset));
  }
    
  }
}

template <typename Dtype>
void SwitchLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const int selector_ind = bottom.size() - 1;
  if(switch_type_==1)
  {
  const Dtype* top_diff = top[0]->gpu_diff();

  CHECK(!propagate_down[selector_ind])<<"Layer cannot backpropagate to selector inputs.";

  for (int n = 0; n < bottom[selector_ind]->num(); n++) {
    int index = static_cast<int>(bottom[selector_ind]->data_at(n, 0 , 0, 0));
    Dtype* bottom_diff = bottom[index]->mutable_gpu_diff();
    caffe_copy(num_elem_, top_diff+top[0]->offset(n),
        bottom_diff + bottom[index]->offset(n));
  }
  }
  else
  {
  CHECK(!propagate_down[0])<<"Bottom layer cannot be propagated to.";

  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  vector<int> count_top_num(top.size());  // keeps count of the top_num

  for (int n = 0; n < bottom[selector_ind]->num(); n++) {
    int index = static_cast<int>(bottom[selector_ind]->data_at(n, 0 , 0, 0));

    const Dtype* top_diff = top[index]->gpu_diff() +
                      top[index]->offset(count_top_num[index]);
    count_top_num[index]++;

    caffe_copy(num_elem_, top_diff, bottom_diff + bottom[0]->offset(n));
  }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SwitchLayer);

}  // namespace caffe
